
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>
#include <ctime>

using namespace std;

__global__
void lista_pierwszych( unsigned long long int *d_sqrtzliczby,
                       unsigned long long int *d_liczba,
                       bool *d_jest){
  long int i = (blockIdx.x*blockDim.x + threadIdx.x);
  if(i<*d_sqrtzliczby+100 && i>2 && i%2!=0){
    if(*d_liczba % i==0){
      *d_jest =1 ;
    }
  }
}

int liczbapierwsza() {
  clock_t begin = clock();
	bool z=1;
	long  y=2;
	int i = 2;
	unsigned long long int x=10000001400000049;
		while (y < sqrt(x)+10) {
			int flag = 0;
			for (i; i <= i / 2; i++){
				printf("1");
				if (y%i == 0){
					flag = 1;
					break;
				}
			}
			if (flag == 0){
				if (x%y == 0) {
					z = 0;
					break;
				}
			}
			y++;
		}
		if (z == 0) {
			printf(" nie jestliczba pierwsza ");
		}
		else
			printf(" jest ");

      clock_t end = clock();
      double elapsed_secs = double(end - begin) / CLOCKS_PER_SEC;
      printf("%f ", elapsed_secs);
	return 1;
}

int main(void){
  liczbapierwsza();
  unsigned long long *liczba, *d_liczba, *sqrtzliczby, *d_sqrtzliczby;
  bool *jest, *d_jest;

  jest= (bool*)malloc(sizeof(bool));
  liczba = (unsigned long long*)malloc(sizeof(unsigned long long));
  sqrtzliczby =(unsigned long long*)malloc(sizeof(unsigned long long));
  hipMalloc(&d_liczba, sizeof(unsigned long long));
  hipMalloc(&d_sqrtzliczby, sizeof(unsigned long long));
  hipMalloc(&d_jest,sizeof(unsigned long long));

  *liczba = 10000001400000049;
  *sqrtzliczby = sqrtl(*liczba);
  printf("%llu ", *sqrtzliczby);
  hipMemcpy(d_liczba,liczba,sizeof(unsigned long long), hipMemcpyHostToDevice);
  hipMemcpy(d_sqrtzliczby,sqrtzliczby, sizeof(unsigned long long),hipMemcpyHostToDevice );
  hipMemcpy(d_jest, jest, sizeof(bool), hipMemcpyHostToDevice);
  clock_t begin = clock();
  if(*liczba%2 ==0){
    printf("%llu ", *liczba);
    printf("nie jest liczba pierwsza");
  }
  else{
    lista_pierwszych<<<((int)sqrt(*liczba)+255)/256, 256>>>( d_sqrtzliczby,d_liczba,d_jest);
    hipMemcpy(jest, d_jest,sizeof(bool),hipMemcpyDeviceToHost);
    if(*jest == 0){
      printf("%llu ", *liczba);
      printf("jest liczba pierwsza");
    }
    else{
      printf("%llu ", *liczba);
      printf("nie jest liczba pierwsza");
    }
  }
  clock_t end = clock();
  double elapsed_secs = double(end - begin) / CLOCKS_PER_SEC;
  printf(" %f ", elapsed_secs);

  hipFree(d_liczba);
  hipFree(d_sqrtzliczby);
  hipFree(d_jest);
  free(jest);
  free(liczba);
  free(sqrtzliczby);
}

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>
#include <ctime>

using namespace std;

__global__
void lista_pierwszych( unsigned long long int *d_sqrtzliczby,
                       unsigned long long int *d_liczba,
                       bool *d_jest,
                       int *current_thread_count,
                       long int q){
  atomicAdd(current_thread_count, 1);
    long int i = (blockIdx.x*blockDim.x + threadIdx.x);
    if((i+61440000*q)<*d_sqrtzliczby+100 && (i+61440000*q)>2 && (i+61440000*q)%2!=0){
      if(*d_liczba % (i+61440000*q)==0){
        *d_jest =1 ;
      }
    }
}


int liczbapierwsza() {
  clock_t begin = clock();
	bool z=1;
	long  y=2;
	int i = 2;
	unsigned long long int x=12808970049658849609;
		while (y < sqrt(x)+10) {
			int flag = 0;
			for (i; i <= i / 2; i++){
				printf("1");
				if (y%i == 0){
					flag = 1;
					break;
				}
			}
			if (flag == 0){
				if (x%y == 0) {
					z = 0;
					break;
				}
			}
			y++;
		}
		if (z == 0) {
			printf("%llu nie jest liczba pierwsza\n ",x);
		}
		else
			printf("%llu jest liczba pierwsza\n ",x);

      clock_t end = clock();
      double elapsed_secs = double(end - begin) / CLOCKS_PER_SEC;
      printf("czas CPU: %f \n", elapsed_secs);
	return 1;
}

int main(void){
  liczbapierwsza();
  unsigned long long *liczba, *d_liczba, *sqrtzliczby, *d_sqrtzliczby;
  bool *jest, *d_jest;
  int tally, *dev_tally;
  hipMalloc((void **)&dev_tally, sizeof(int));
  tally = 0;
  hipMemcpy(dev_tally, &tally, sizeof(int), hipMemcpyHostToDevice);
  jest= (bool*)malloc(sizeof(bool));
  liczba = (unsigned long long*)malloc(sizeof(unsigned long long));
  sqrtzliczby =(unsigned long long*)malloc(sizeof(unsigned long long));
  hipMalloc(&d_liczba, sizeof(unsigned long long));
  hipMalloc(&d_sqrtzliczby, sizeof(unsigned long long));
  hipMalloc(&d_jest,sizeof(unsigned long long));

  *liczba = 12808970049658849609;
  *sqrtzliczby = sqrtl(*liczba);
  long int op= (int)ceil(*sqrtzliczby/61440000);
  printf("\n %ld \n", op);
  printf("pierwiastek z liczby: %llu\n ", *sqrtzliczby);
  hipMemcpy(d_liczba,liczba,sizeof(unsigned long long), hipMemcpyHostToDevice);
  hipMemcpy(d_sqrtzliczby,sqrtzliczby, sizeof(unsigned long long),hipMemcpyHostToDevice );
  hipMemcpy(d_jest, jest, sizeof(bool), hipMemcpyHostToDevice);

  clock_t begin = clock();
  if(*liczba%2 ==0){
    printf("%llu nie jest liczba pierwsza\n", *liczba);
  }
  else{
    for(int q=0;q <= op; q++){
      lista_pierwszych<<<60000,1024>>>( d_sqrtzliczby,d_liczba,d_jest, dev_tally ,q);
    }
      hipMemcpy(jest, d_jest,sizeof(bool),hipMemcpyDeviceToHost);
      hipMemcpy(&tally, dev_tally, sizeof(int), hipMemcpyDeviceToHost);
    printf("total number of threads that executed was: %d\n", tally);
    if(*jest == 0){
      printf("%llu jest liczba pierwsza\n", *liczba);
    }
    else{
      printf("%llu nie jest liczba pierwsza\n", *liczba);
    }
  }
  clock_t end = clock();
  double elapsed_secs = double(end - begin) / CLOCKS_PER_SEC;
  printf("czas GPU: %f\n ", elapsed_secs);

  hipFree(d_liczba);
  hipFree(d_sqrtzliczby);
  hipFree(d_jest);
  free(jest);
  free(liczba);
  free(sqrtzliczby);
}

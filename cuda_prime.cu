
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>
#include <ctime>

using namespace std;

__global__
void lista_pierwszych( unsigned long long int *d_sqrtzliczby,
                       unsigned long long int *d_liczba,
                       bool *d_jest,
                       int *current_thread_count){
  atomicAdd(current_thread_count, 1);
  long int i = (blockIdx.x*blockDim.x + threadIdx.x);
  if(i<*d_sqrtzliczby+100 && i>2 && i%2!=0){
    if(*d_liczba % i==0){
      *d_jest =1 ;
    }
  }
}

__device__ int getGlobalIdx_1D_1D(){
	return blockIdx.x *blockDim.x + threadIdx.x;
}

int liczbapierwsza() {
  clock_t begin = clock();
	bool z=1;
	long  y=2;
	int i = 2;
	unsigned long long int x=281200132374529;
		while (y < sqrt(x)+10) {
			int flag = 0;
			for (i; i <= i / 2; i++){
				printf("1");
				if (y%i == 0){
					flag = 1;
					break;
				}
			}
			if (flag == 0){
				if (x%y == 0) {
					z = 0;
					break;
				}
			}
			y++;
		}
		if (z == 0) {
			printf(" nie jestliczba pierwsza ");
		}
		else
			printf(" jest ");

      clock_t end = clock();
      double elapsed_secs = double(end - begin) / CLOCKS_PER_SEC;
      printf("%f ", elapsed_secs);
	return 1;
}

int main(void){
  liczbapierwsza();
  unsigned long long *liczba, *d_liczba, *sqrtzliczby, *d_sqrtzliczby;
  bool *jest, *d_jest;
  int tally, *dev_tally;
  hipMalloc((void **)&dev_tally, sizeof(int));
  tally = 0;
  hipMemcpy(dev_tally, &tally, sizeof(int), hipMemcpyHostToDevice);
  jest= (bool*)malloc(sizeof(bool));
  liczba = (unsigned long long*)malloc(sizeof(unsigned long long));
  sqrtzliczby =(unsigned long long*)malloc(sizeof(unsigned long long));
  hipMalloc(&d_liczba, sizeof(unsigned long long));
  hipMalloc(&d_sqrtzliczby, sizeof(unsigned long long));
  hipMalloc(&d_jest,sizeof(unsigned long long));

  *liczba = 281200132374529;
  *sqrtzliczby = sqrtl(*liczba);
  printf("%llu ", *sqrtzliczby);
  hipMemcpy(d_liczba,liczba,sizeof(unsigned long long), hipMemcpyHostToDevice);
  hipMemcpy(d_sqrtzliczby,sqrtzliczby, sizeof(unsigned long long),hipMemcpyHostToDevice );
  hipMemcpy(d_jest, jest, sizeof(bool), hipMemcpyHostToDevice);

  clock_t begin = clock();
  if(*liczba%2 ==0){
    printf("%llu ", *liczba);
    printf("nie jest liczba pierwsza");
  }
  else{
    lista_pierwszych<<<((int)sqrt(*liczba)+255)/256, 256>>>( d_sqrtzliczby,d_liczba,d_jest, dev_tally);
    hipMemcpy(jest, d_jest,sizeof(bool),hipMemcpyDeviceToHost);
    hipMemcpy(&tally, dev_tally, sizeof(int), hipMemcpyDeviceToHost);
    printf("total number of threads that executed was: %d\n", tally);
    if(*jest == 0){
      printf("%llu ", *liczba);
      printf("jest liczba pierwsza");
    }
    else{
      printf("%llu ", *liczba);
      printf("nie jest liczba pierwsza");
    }
  }
  clock_t end = clock();
  double elapsed_secs = double(end - begin) / CLOCKS_PER_SEC;
  printf(" %f ", elapsed_secs);

  hipFree(d_liczba);
  hipFree(d_sqrtzliczby);
  hipFree(d_jest);
  free(jest);
  free(liczba);
  free(sqrtzliczby);
}
